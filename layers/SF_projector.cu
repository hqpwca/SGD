#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <random>
#include <algorithm>

#include "omp.h" 
#include <hip/hip_runtime.h>

#include "SF_projector.hh"
#include "../utils/exception.hh"

template <class T>
__device__ static void sort2(T* a, T* b)
{
    if (*a > *b)
    {
        T tmp = *b;
        *b = *a;
        *a = tmp;
    }
}

template <class T>
__device__ static void sort4(T* a, T* b, T* c, T* d)
{

    sort2<T>(a,b);
    sort2<T>(c,d);
    sort2<T>(a,c);
    sort2<T>(b,d);
    sort2<T>(b,c);

    // sort3
    // sort2(b, c);
    // sort2(a, b);
    // sort2(b, c);

}

__device__ static void gamma_calculate(float s1, float s2, float *us, float *gamma) {
    float tmp = 0.0f;
    float b1, b2;

    b1 = fmaxf(s1, us[0]);
    b2 = fminf(s2, us[1]);

    if(b2 > b1){
        float _a = b1 - us[0];
        float _b = b2 - b1;
        tmp =  (_b * _b + 2 * _a * _b) / (2 * (us[1] - us[0]));
    }
    else {
        tmp = 0.0f;
    }
    if(tmp == tmp) gamma[0] += tmp;

    b1 = fmaxf(s1, us[1]);
    b2 = fminf(s2, us[2]);

    if(b2 > b1){
        tmp = b2 - b1;
    }
    else{
        tmp = 0.0f;
    }
    if(tmp == tmp) gamma[0] += tmp;

    b1 = fmaxf(s1, us[2]);
    b2 = fminf(s2, us[3]);

    if(b2 > b1){
        float _b = b2 - b1;
        float _c = us[3] - b2;
        tmp = (_b * _b + 2 * _b * _c) / (2 * (us[3] - us[2]));
    }
    else{
        tmp = 0.0f;
    }
    if(tmp == tmp) gamma[0] += tmp;
}

// block_size: (8, 8, 64) ILP on z-axis.

template <class T>
__global__ void SF_project(T proj, const float *vol, int3 n3xyz, double3 d3xyz, const float *pm, int nu, int nv, double3 src, double rect_rect_factor, int z_size, bool trap_v = false)
{
    int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iy = (blockIdx.y * blockDim.y) + threadIdx.y;
    int oz = (blockIdx.z * blockDim.z) + threadIdx.z;

    int z_start = oz * z_size;
    int z_end = z_start + z_size;
    z_end = min(n3xyz.z, z_end);

    if(ix >= n3xyz.x || iy >= n3xyz.y || z_end <= z_start) return;

    int nx,ny,nz;
    float min_u, max_u, min_v, max_v;
    float s1, s2;
    float us[4] = {0.0};
    float vs[4] = {0.0};
    int idxv;
    float C;

    nx = n3xyz.x, ny = n3xyz.y, nz = n3xyz.z;
    nx = nx/2; ny = ny/2; nz = nz/2;

    float weight, signy1, signy2, signx1, signx2, signz1, signz2;
    unsigned int nuv = nu*nv;
    size_t idx, idx0;

    idx0 = (iy*n3xyz.x) + ix;
        
    float pmv2, pmv3;
    float u1, u2, u3, u4, u5, u6, u7, u8;
    float v1, v2, v3, v4;
    pmv2 = pm[4]*ix + pm[5]*iy + pm[7]; //matrix multiplication result without z-axis
    pmv3 = pm[8]*ix + pm[9]*iy + pm[11]; //matrix multiplication result without z-axis, normalization

    signy1 = iy - 0.5f;
    signy2 = iy + 0.5f;
    signx1 = ix - 0.5f;
    signx2 = ix + 0.5f;

    //matrix multiplication result without z-axis for 4 corners and normalization coefficients
    u1 = pm[0]*signx1 + pm[1]*signy1 + pm[3];
    u2 = pm[8]*signx1 + pm[9]*signy1 + pm[11];

    u3 = pm[0]*signx2 + pm[1]*signy1 + pm[3];
    u4 = pm[8]*signx2 + pm[9]*signy1 + pm[11];

    u5 = pm[0]*signx1 + pm[1]*signy2 + pm[3];
    u6 = pm[8]*signx1 + pm[9]*signy2 + pm[11];

    u7 = pm[0]*signx2 + pm[1]*signy2 + pm[3];
    u8 = pm[8]*signx2 + pm[9]*signy2 + pm[11];

    if (trap_v) {
        v1 = pm[4]*signx1 + pm[5]*iy + pm[7];
        v2 = pm[4]*signx2 + pm[5]*iy + pm[7];
        v3 = pm[8]*signx1 + pm[9]*iy + pm[11];
        v4 = pm[8]*signx2 + pm[9]*iy + pm[11];
    }

    for (int iz = z_start; iz < z_end; ++iz) {
        idx = ( (size_t) (iz) )*( (size_t) n3xyz.x*n3xyz.y ) + idx0;

        signz1 = (iz-0.5f);
        signz2 = (iz+0.5f);

        if(!trap_v) {
            vs[0] = ( pmv2 + pm[6] *signz1 ) / ( pmv3 + pm[10]*signz1 );

            min_v = ceilf(vs[0] - 0.5f);
            if ( min_v >= nv ) return;

            vs[1] = ( pmv2 + pm[6] *signz2 ) / ( pmv3 + pm[10]*signz2 );
            
            max_v = ceilf(vs[1] - 0.5f);
            if ( max_v < 0 ) continue;
        }
        else {
            vs[0] = ( v1 + pm[6] * signz1 ) / ( v3 + pm[10] * signz1 );
            vs[1] = ( v1 + pm[6] * signz2 ) / ( v3 + pm[10] * signz2 );
            vs[2] = ( v2 + pm[6] * signz1 ) / ( v4 + pm[10] * signz1 );
            vs[3] = ( v2 + pm[6] * signz2 ) / ( v4 + pm[10] * signz2 );

            sort4<float>(vs, vs+1, vs+2, vs+3);

            min_v = ceilf(vs[0] - 0.5f);
            max_v = ceilf(vs[3] - 0.5f);

            if ( ( max_v < 0 ) || ( min_v >= nv ) ) continue;
        }

        C = vol[idx];
        if (C == 0) continue;

        weight = rsqrtf( (d3xyz.x*(ix - nx)-src.x)*(d3xyz.x*(ix - nx)-src.x) + (d3xyz.y*(iy - ny)-src.y)*(d3xyz.y*(iy - ny)-src.y) + (d3xyz.z*(iz - nz)-src.z)*(d3xyz.z*(iz - nz)-src.z) );
        weight *= weight;

        us[0] = ( u1 + pm[2]*iz ) / ( u2 + pm[10]*iz );

        us[1] = ( u3 + pm[2]*iz ) / ( u4 + pm[10]*iz );

        us[2] = ( u5 + pm[2]*iz ) / ( u6 + pm[10]*iz );

        us[3] = ( u7 + pm[2]*iz ) / ( u8 + pm[10]*iz );

        sort4<float>(us, us+1, us+2, us+3);

        min_u = ceilf(us[0] - 0.5f);
        max_u = ceilf(us[3] - 0.5f);
        
        if ( ( max_u < 0 ) || ( min_u >= nu ) ) continue;
        
        if(trap_v){
            C *= weight * rect_rect_factor * (2 / ( ((us[3]-us[0])+(us[2]-us[1])) )) * (2 / ( vs[3] - vs[0] ) + (vs[2] - vs[1]) );
        }
        else {
            C *= weight * rect_rect_factor * (2 / ( ((us[3]-us[0])+(us[2]-us[1])) )) * (1 / ( vs[1] - vs[0] ) );
        }

        for (int ti = 0; ti < max_u - min_u + 1; ++ti) {
            int i = ti + min_u;

            s1 = i - 0.5f;
            s2 = i + 0.5f;

            float gamma = 0.0f;
            gamma_calculate(s1, s2, us, &gamma);

            for (int tj = 0; tj < max_v - min_v + 1; ++tj) {
                int j = tj + min_v;
                s1 = j + 0.5f;
                s2 = j - 0.5f;

                float f2;

                if(trap_v) {
                    float gamma2 = 0.0f;
                    gamma_calculate(s2, s1, vs, &gamma2);

                    f2 = gamma * gamma2;
                }
                else
                    f2 = gamma * fmaxf(fminf(s1,vs[1]) - fmaxf(s2,vs[0]),0);

                idxv = j * nu + i;
                
                if(idxv < nuv && idxv >= 0 && f2 == f2) {
                    atomicAdd(proj+idxv, f2);
                }
            }
        }
    }
}

// block_size: (8, 8, 64) ILP on z-axis.
template <class T>
__global__ void SF_backproject(T proj, float *vol, int3 n3xyz, double3 d3xyz, const float *pm, int nu, int nv, double3 src, double rect_rect_factor, int z_size)
{
    int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iy = (blockIdx.y * blockDim.y) + threadIdx.y;
    int oz = (blockIdx.z * blockDim.z) + threadIdx.z;

    int z_start = oz * z_size;
    int z_end = z_start + z_size;
    z_end = min(n3xyz.z, z_end);

    int nx,ny,nz;
    float min_u, max_u, min_v, max_v;
    float s1, s2;
    float us[4] = {0.0};
    float vs[2] = {0.0};
    int idxv;
    float C;

    nx = n3xyz.x, ny = n3xyz.y, nz = n3xyz.z;
    nx = nx/2; ny = ny/2; nz = nz/2;

    float weight, signy1, signy2, signx1, signx2, signz1, signz2;
    unsigned int nuv = nu*nv;
    size_t idx, idx0;

    idx0 = (iy*n3xyz.x) + ix;
        
    float pmv2, pmv3;
    float u1, u2, u3, u4, u5, u6, u7, u8;
    pmv2 = pm[4]*ix + pm[5]*iy + pm[7]; //matrix multiplication result without z-axis
    pmv3 = pm[8]*ix + pm[9]*iy + pm[11]; //matrix multiplication result without z-axis, normalization

    signy1 = iy - 0.5f;
    signy2 = iy + 0.5f;
    signx1 = ix - 0.5f;
    signx2 = ix + 0.5f;

    //matrix multiplication result without z-axis for 4 corners and normalization coefficients
    u1 = pm[0]*signx1 + pm[1]*signy1 + pm[3];
    u2 = pm[8]*signx1 + pm[9]*signy1 + pm[11];

    u3 = pm[0]*signx2 + pm[1]*signy1 + pm[3];
    u4 = pm[8]*signx2 + pm[9]*signy1 + pm[11];

    u5 = pm[0]*signx1 + pm[1]*signy2 + pm[3];
    u6 = pm[8]*signx1 + pm[9]*signy2 + pm[11];

    u7 = pm[0]*signx2 + pm[1]*signy2 + pm[3];
    u8 = pm[8]*signx2 + pm[9]*signy2 + pm[11];

    for (int iz = z_start; iz < z_end; ++iz) {
        idx = ( (size_t) (iz) )*( (size_t) n3xyz.x*n3xyz.y ) + idx0;

        signz1 = (iz-0.5f);
        signz2 = (iz+0.5f);

        vs[0] = ( pmv2 + pm[6] *signz1 ) / ( pmv3 + pm[10]*signz1 );

        min_v = ceilf(vs[0] - 0.5f);
        if ( min_v >= nv ) return;

        vs[1] = ( pmv2 + pm[6] *signz2 ) / ( pmv3 + pm[10]*signz2 );
        
        max_v = ceilf(vs[1] - 0.5f);
        if ( max_v < 0 ) continue;

        weight = rsqrtf( (d3xyz.x*(ix - nx)-src.x)*(d3xyz.x*(ix - nx)-src.x) + (d3xyz.y*(iy - ny)-src.y)*(d3xyz.y*(iy - ny)-src.y) + (d3xyz.z*(iz - nz)-src.z)*(d3xyz.z*(iz - nz)-src.z) );

        us[0] = ( u1 + pm[2]*iz ) / ( u2 + pm[10]*iz );

        us[1] = ( u3 + pm[2]*iz ) / ( u4 + pm[10]*iz );

        us[2] = ( u5 + pm[2]*iz ) / ( u6 + pm[10]*iz );

        us[3] = ( u7 + pm[2]*iz ) / ( u8 + pm[10]*iz );

        sort4<float>(us, us+1, us+2, us+3);

        min_u = ceilf(us[0] - 0.5f);
        max_u = ceilf(us[3] - 0.5f);
        
        if ( ( max_u < 0 ) || ( min_u >= nu ) ) continue;
        
        C = weight * rect_rect_factor * (2 / ( ((us[3]-us[0])+(us[2]-us[1])) )) * (1 / ( vs[1] - vs[0] ) );

        float sumV = 0.0f;

        for (int ti = 0; ti < max_u - min_u + 1; ++ti) {
            int i = ti + min_u;

            s1 = i - 0.5f;
            s2 = i + 0.5f;
            float gamma = 0.0f;

            gamma_calculate(s1, s2, us, &gamma);

            for (int tj = 0; tj < max_v - min_v + 1; ++tj) {
                int j = tj + min_v;
                s1 = j + 0.5f;
                s2 = j - 0.5f;

                float f = gamma * fmaxf(fminf(s1,vs[1]) - fmaxf(s2,vs[0]),0);

                idxv = j * nu + i;
                
                if(idxv < nuv && idxv >= 0) {
                    sumV += proj[idxv] * f;
                }
            }
        }

        sumV *= C;
        
        if (sumV == sumV){
            atomicAdd(&vol[idx], sumV);
        }

    }
}


void SF::project(Matrix &vol, Matrix &proj, double weight, bool tt) { // data processing on device
    for(int p=0; p<geodata->np; p++) {
        //if(p != 0 && p != 15 && p != 35 && p != 45) continue;

        float lsd = *geodata->lsds[p];
        double factor = lsd * lsd * geodata->dxyz.y * geodata->dxyz.z / (geodata->duv.x * geodata->duv.y);
        //std::cout << p <<' ' << p * geodata->nuv.x * geodata->nuv.y << std::endl;

        SF_project<float *> <<<vgrid, vblock>>>(proj(p * geodata->nuv.x * geodata->nuv.y), vol(0), geodata->nxyz, geodata->dxyz, geodata->pmis(p*12), geodata->nuv.x, geodata->nuv.y,
                                      make_double3(*geodata->srcs[p*3], *geodata->srcs[p*3+1], *geodata->srcs[p*3+2]), factor, Z_SIZE, tt);
        hipDeviceSynchronize();

        proj.copyDeviceToHost();
    }
}

void SF::project(Matrix &vol, MatrixD &proj, double weight, bool tt) { // data processing on device
    for(int p=0; p<geodata->np; p++) {
        if(p != 0 && p != 15 && p != 35 && p != 45) continue;

        float lsd = *geodata->lsds[p];
        double factor = lsd * lsd * geodata->dxyz.y * geodata->dxyz.z / (geodata->duv.x * geodata->duv.y);
        //std::cout << p <<' ' << p * geodata->nuv.x * geodata->nuv.y << std::endl;

        SF_project<double *> <<<vgrid, vblock>>>(proj(p * geodata->nuv.x * geodata->nuv.y), vol(0), geodata->nxyz, geodata->dxyz, geodata->pmis(p*12), geodata->nuv.x, geodata->nuv.y,
                                      make_double3(*geodata->srcs[p*3], *geodata->srcs[p*3+1], *geodata->srcs[p*3+2]), factor, Z_SIZE, tt);
        hipDeviceSynchronize();

        proj.copyDeviceToHost();
    }
}

void SF::back_project(Matrix &vol, Matrix &proj, double weight, bool tt) {
    for(int p=0; p<geodata->np; p++) {
        float lsd = *geodata->lsds[p];
        double factor = lsd  * geodata->dxyz.x / (geodata->np);

        SF_backproject<float *> <<<vgrid, vblock>>>(proj(p * geodata->nuv.x * geodata->nuv.y), vol(0), geodata->nxyz, geodata->dxyz, geodata->pmis(p*12), geodata->nuv.x, geodata->nuv.y,
                                      make_double3(*geodata->srcs[p*3], *geodata->srcs[p*3+1], *geodata->srcs[p*3+2]), factor, Z_SIZE);
        hipDeviceSynchronize();
    }
}

Matrix& SF::forward(hipblasHandle_t &cublasH, Matrix &x) {
    Matrix *y = new Matrix(geodata->nuv.x * geodata->nuv.y, geodata->np);
    y->allocateCudaMemory();
    project(x, *y, 1.0f);

    return *y;
}

Matrix& SF::back_prop(hipblasHandle_t &cublasH, Matrix &od, float lr) {
    
}

SF::SF(GeoData *geo) {
    geodata = geo;

    int bx = (geo->nxyz.x + 7) / 8;
    int by = (geo->nxyz.y + 7) / 8;
    int bz = (geo->nxyz.z + 63) / 64;

    vblock = dim3(8, 8, 1);
    vgrid = dim3(bx, by, bz);
}

SF::~SF() {
    
}

//Test for only project/backproject kernel;