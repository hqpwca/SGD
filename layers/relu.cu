#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <random>

#include "relu.hh"
#include "../utils/exception.hh"

ReLU:ReLU() { }

ReLU:~ReLU() { }

Matrix& ReLU::forward(hipblasHandle_t &cublasH, Matrix &x) { //(_, batch_size)
    this.X = x;
    Y.allocateMemoryIfNotAllocated(x.shape);

    CUBLAS_CHECK(hipblasIsamax(cublasH, ));

    return Y;
}

Matrix& ReLU::back_prop(hipblasHandle_t &cublasH, Matrix &od) { //(_, batch_size)

    return d;
}