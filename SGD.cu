/*
    A program trying to implement SGD using CUDA.
    Target function: Y = F(X) = (x1^t1 + ((x1+x2)/2)^t2 + ... + ((x1+...+x16)/16)^t16)/16 (t_i \in [0.5, 2]) (x_i \in [0, 1])
    Network Structure: Input -- Linear(16, 32) -- ReLU -- Linear(32, 16) -- ReLU -- Linear(16, 1) -- Sigmoid
*/

#include <algorithm>
#include <cmath>
#include <iostream>
#include <random>
#include <exception>

#include "assert.h"
#include "stdlib.h"

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "network.hh"

hipblasHandle_t handle;

Matrix calcLoss(Matrix &batch_output, Matrix &network_output)
{
    
}

void init_network(Network &N) {
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    N.addLayer(new Linear(Shape(16, 32)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(32, 16)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(16, 1)));
    N.addLayer(new Sigmoid());
}

void cleanup() {
    hipblasDestroy(handle);
}

void train(float lr = 0.1) {

}

float test() {

}

int main() {

    Network N;
    init_network(N);

    train();

    cleanup();

    return 0;
}