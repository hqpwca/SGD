#include "hip/hip_runtime.h"
/*
    A program trying to implement SGD using CUDA.
    Target function: Y = F(X) = (x1^t1 + ((x1+x2)/2)^t2 + ... + ((x1+...+x16)/16)^t16)/16 (t_i \in [0.5, 2]) (x_i \in [0, 1])
    Network Structure: Input -- Linear(16, 32) -- ReLU -- Linear(32, 16) -- ReLU -- Linear(16, 1) -- Sigmoid
*/

#include <algorithm>
#include <cmath>
#include <iostream>
#include <random>
#include <exception>

#include "assert.h"
#include "stdlib.h"

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "network.hh"
#include "utils/dataset.hh"

hipblasHandle_t handle;
std::vector<float> T;
Dataset *train, *test;

Matrix calcSquareLoss(Matrix &batch_output, Matrix &network_output) // (1, batch_size)
{
    assert(batch_output.shape.x == network_output.shape.x && batch_output.shape.y == network_output.shape.y);

    Matrix m(batch_output.shape);
    for (int i = 0; i < batch_output.shape.x * batch_output.shape.y; ++ i) {
        m[i] =  std::pow((network_output[i] - batch_output[i]), 2.0);
    }
    return m;
}

float sumLoss(Matrix &m) {
    float sum = 0.0;
    for (int i = 0; i < m.shape.x * m.shape.y; ++ i) {
        sum += m[i];
    }
    return sum;
}

void init_network(Network &N) {
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    N.addLayer(new Linear(Shape(16, 32)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(32, 16)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(16, 1)));
    N.addLayer(new Sigmoid());
}

void cleanup() {
    hipblasDestroy(handle);
    delete train;
    delete test;
}

void train_SGD(Network N, Dataset *d, float lr = 0.1, int epoches = 100) {
    for (int epoch = 0; epoch < epoches; ++epoch) {
        d->nextEpoch();

        int num_batches = -1;
        float sumloss = 0.0;
        while(d->nextBatch()) {
            ++ num_batches;
            Matrix input = d->getBatchInput();
            Matrix output = d->getBatchOutput();

            //Forward
            Matrix net_output = N.forward(handle, input);
            net_output.copyDeviceToHost();

            //Calc loss
            Matrix loss = calcSquareLoss(output, net_output);
            sumloss += sumLoss(loss);
            loss.copyHostToDevice();

            //Back prop
            N.back_prop(handle, loss, lr);
        }
        std::cerr << "Finished Epoch #" << epoch << " Loss: " << sumloss / num_batches / 32 << std::endl;
    }
}

float test_SGD(Network N, Dataset *d) {

}

int main(int argc, char *argv[]) {
    Network N;
    init_network(N);

    train = new Dataset(T, 16, 1, 32, 2048);
    test = new Dataset(T, 16, 1, 32, 512);

    train_SGD(N, train);

    cleanup();

    return 0;
}