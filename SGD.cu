#include "hip/hip_runtime.h"
/*
    A program trying to implement SGD using CUDA.
    Target function: Y = F(X) = (x1^t1 + ((x1+x2)/2)^t2 + ... + ((x1+...+x16)/16)^t16)/16 (t_i \in [0.5, 2]) (x_i \in [0, 1])
    Network Structure: Input -- Linear(16, 32) -- ReLU -- Linear(32, 16) -- ReLU -- Linear(16, 1) -- Sigmoid
*/

#include <algorithm>
#include <cmath>
#include <iostream>
#include <random>
#include <exception>

#include "assert.h"
#include "stdlib.h"

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "network.hh"
#include "utils/dataset.hh"

#define BATCH_SIZE 256

hipblasHandle_t handle;
std::vector<float> T;
Dataset *train, *val, *test;

Matrix calcSquareLoss(Matrix &batch_output, Matrix &network_output) // (1, batch_size)
{
    assert(batch_output.shape.x == network_output.shape.x && batch_output.shape.y == network_output.shape.y);

    Matrix m(batch_output.shape);
    m.allocateMemory();
    for (int i = 0; i < batch_output.shape.x * batch_output.shape.y; ++ i) {
        m[i] =  std::pow((network_output[i] - batch_output[i]), 2.0);
    }
    return m;
}

float sumLoss(Matrix &m) {
    float sum = 0.0;
    for (int i = 0; i < m.shape.x * m.shape.y; ++ i) {
        sum += m[i];
    }
    return sum;
}

void init_network(Network &N) {
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    N.addLayer(new Linear(Shape(16, 16)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(16, 32)));
    N.addLayer(new ReLU());
    N.addLayer(new Linear(Shape(32, 16)));
    N.addLayer(new Sigmoid());
    N.addLayer(new Linear(Shape(16, 1)));
}

void cleanup() {
    hipblasDestroy(handle);
    delete train;
    delete test;
}

void train_SGD(Network &N, Dataset *train, Dataset *val, float lr = 0.1, int epoches = 100, int batch_size = BATCH_SIZE) {
    for (int epoch = 0; epoch < epoches; ++epoch) {
        train->nextEpoch();

        int num_batches = -1;
        float sumloss = 0.0;
        while(train->nextBatch()) {
            ++ num_batches;
            Matrix input = train->getBatchInput();
            Matrix output = train->getBatchOutput();

            //Forward
            Matrix net_output = N.forward(handle, input);
            net_output.copyDeviceToHost();

            //Calc loss
            Matrix loss = calcSquareLoss(output, net_output);
            sumloss += sumLoss(loss);
            loss.copyHostToDevice();

            //Back prop
            N.back_prop(handle, loss, lr*10/(10+epoch)); //Dynamic Learning Rate
        }

        std::cerr << "Finished Epoch #" << epoch+1 << " Train Loss: " << sumloss / num_batches / batch_size;

        //Validate
        val->nextEpoch();

        num_batches = -1;
        float sumval = 0.0;
        while(val->nextBatch()) {
            ++ num_batches;
            Matrix input = val->getBatchInput();
            Matrix output = val->getBatchOutput();

            //Forward
            Matrix net_output = N.forward(handle, input);
            net_output.copyDeviceToHost();

            //Calc loss
            Matrix loss = calcSquareLoss(output, net_output);
            sumval += sumLoss(loss);
        }
        std::cerr << " Valid Loss: " << sumval / num_batches / batch_size << std::endl;
    }
}

void validate_SGD(Network &N, Dataset *d) {

}

void test_SGD(Network &N, Dataset *d) {
    d->nextEpoch();

    int num_batches = -1;
    float sumloss = 0.0;
    while(d->nextBatch()) {
        ++ num_batches;
        Matrix input = d->getBatchInput();
        Matrix output = d->getBatchOutput();

        //Forward
        Matrix net_output = N.forward(handle, input);
        net_output.copyDeviceToHost();

        //Calc loss
        Matrix loss = calcSquareLoss(output, net_output);
        sumloss += sumLoss(loss);
    }

    std::cerr << "Finished Epoch #" << "Test" << " Loss: " << sumloss / num_batches / BATCH_SIZE << std::endl;
}

int main(int argc, char *argv[]) {
    Network N;
    init_network(N);

    train = new Dataset(T, 16, 1, BATCH_SIZE, 4096);
    val = new Dataset(T, 16, 1, BATCH_SIZE, 512);
    test = new Dataset(T, 16, 1, BATCH_SIZE, 512);

    train_SGD(N, train, val, 0.05);

    test_SGD(N, test);

    cleanup();

    return 0;
}